#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include ""

//#define block 1
//#define threads 17

//!!!! logo provlhmatwn to programma douleuei mono me block * thread >= apo auto pou upologizei
//giati se ka8e thread dinei 1 i tou loop pou kanei parallhla

int test(int *a, int n);
int parsetable(int *a, int n);

__global__ void parallel(int elements_table, int *a, int *count,int max) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	if (index < elements_table) {
		atomicAdd(count + a[index], 1);
	}
	__syncthreads();
}

__global__ void nestedparallel(int max, int *temp, int *count, int value) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int i = index;
	if (i < max + 1 && i != 0) {
		if (i < value)
			temp[i] = count[i];
		else 
			temp[i] = count[i] + count[i - value];
	}
	__syncthreads();

	if (index < max+1)
		count[index] = temp[index];
	__syncthreads();
}


int main(int argc, char *argv[]) {

	int i, counter, value, j, max = 0;
	int* a;
	int *d_a, *d_count, *d_temp;
	double delta, start, end;

	if (argc != 4) {
		printf("No� correct number of arguments\n");
		return 1;
	}
    //4 arguments to elements table , ta blocks kai ta thread 
	int elements_table = strtol(argv[1], NULL, 10);
	int block = strtol(argv[2], NULL, 10);
    int threads = strtol(argv[2], NULL, 10);

	a = (int *)malloc(elements_table * sizeof(int));
	hipMalloc(&d_a, elements_table * sizeof(float));

	hipGetDeviceCount(&counter);
	printf("There are %d GPU devices in your system\n", counter);

	for (i = 0; i < elements_table; i++)
	{
		int x = rand() % elements_table;
		if (x == 0)
			x = x + (rand() % elements_table) + 1;
		a[i] = x;
		if (a[i] > max)
			max = a[i];
	}

	int *count = (int *)malloc((max + 1) * sizeof(int));
	int *temp = (int *)malloc((max + 1) * sizeof(int));
	int *output = (int *)malloc(elements_table * sizeof(int));

	hipMalloc(&d_temp, (max + 1) * sizeof(int));
	hipMalloc(&d_count, (max + 1) * sizeof(int));

	for (i = 0; i < max + 1; i++) {
		count[i] = 0;
		temp[i] = 0;
	}

	//parallhlopoihshmh perioxh 1h

	hipMemcpy(d_a, a, elements_table * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_count, count, (max + 1) * sizeof(int), hipMemcpyHostToDevice);

	parallel << <block, threads >> > (elements_table, d_a, d_count, max+1);

	hipMemcpy(count, d_count, (max + 1) * sizeof(int), hipMemcpyDeviceToHost);

	//to delta einai log me bash to 2 tou n
	// gia ta log kai pow prepei otan kanoume gcc na valoume k -lm 
	// gia thn math.h sto telos  
	delta = (log(elements_table) / log(2));
	int e = (int)delta;

	//2h parallel zone

	hipMemcpy(d_temp, temp, (max + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_count, count, (max + 1) * sizeof(int), hipMemcpyHostToDevice);

	for (j = 0; j < e + 1; j++) {
		value = pow(2, j);
		nestedparallel <<<block, threads>> > (max, d_temp, d_count, value);
	}

	hipMemcpy(count, d_count, (max + 1) * sizeof(int), hipMemcpyDeviceToHost);

	//telos perioxhs
	for (i = 0; i < elements_table; i++) {
		output[count[a[i]] - 1] = a[i];
		count[a[i]]--;
	}

	for (i = 0; i < elements_table; i++)
		a[i] = output[i];
	parsetable(a,elements_table);

	test(a, elements_table);

    hipFree(d_count);
	hipFree(d_a);
	hipFree(d_temp);
	return 0;

}

int test(int *a, int n) {
	// if 0 == false
	int pass = 1;
	for (int i = 1; i < n; i++)
	{
		if (a[i] < a[i - 1])
			pass = 0;
	}
	if (pass)
		printf("The list is sorted\n");
	else
		printf("The list isn't sorted\n");
	return 0;
}

int parsetable(int *a, int n) {

	printf("\n");
	for (int i = 0; i < n; i++)
		printf("-%d-", a[i]);
	printf("\n");
	return 0;
}
