#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


#define N 128
#define base 0
//sto visual studio ta kanw define otan ta kanw compile ta dinw orismata
//#define block_count 100;
//#define thread_count 100;

//!!!! logo provlhmatwn to programma douleuei mono me block * thread >= apo auto pou upologizei
//giati se ka8e thread dinei 1 i tou loop pou kanei parallhla

__global__ void parallelf(char *buffer,int *freq, long file_size) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	
    /*epeidh egrafa se visual studio kai eixe kapoio provlhma me pathing den katafera na 
	testarw to 
	for (int i = index; i < file_size;i= i + blockDim.x * gridDim.x); anti gia to if
	pou 8ewritika 8a to ekane na trexei kai se ligotera apo ton ari8mo file_size
	block kai thread dinontas ksana sto ka8e thread afou teleiwne kai allo workload
	*/
	if (index < (int)file_size ) {
		atomicAdd(freq + buffer[index], 1);
	}
	__syncthreads();

}

int main(int argc, char *argv[]) {

	FILE *pFile;
	long file_size;
	char * buffer;
	char *d_buffer;
	int	*d_freq;
	const char * filename;
	size_t result;
	int j, freq[N],counter;
	int thread_count;
	int block_count;

	if (argc != 4) {
		printf("Usage : %s <file_name>\n", argv[0]);
		return 1;
	}
	filename = argv[1];
	thread_count = strtol(argv[2], NULL, 10);
	block_count = strtol(argv[2], NULL, 10);
	pFile = fopen(filename, "rb");
	if (pFile == NULL) { perror("File error\n"); return 2; }

	hipGetDeviceCount(&counter);
	printf("There are %d GPU devices in your system\n", counter);

	// obtain file size:
	fseek(pFile, 0, SEEK_END);
	file_size = ftell(pFile);
	rewind(pFile);
	printf("file size is %ld\n", file_size);

	// allocate memory to contain the file:
	buffer = (char*)malloc(sizeof(char)*file_size);

	hipMalloc(&d_buffer, (sizeof(char)*file_size));
	hipMalloc(&d_freq, N * sizeof(int));

	if (buffer == NULL) { printf("Memory error\n"); return 3; }

	// copy the file into the buffer:
	result = fread(buffer, 1, file_size, pFile);
	if (result != file_size) { printf("Reading error\n"); return 4; }
	//gemizei 128 8eseis me 0
	for (j = 0; j < N; j++) {
		freq[j] = 0;
	}
	//auto p 8eloume parallhlo , gemizei to freq 
	//parallhlopoihshmh perioxh 

	hipMemcpy(d_buffer, buffer, (sizeof(char)*file_size), hipMemcpyHostToDevice);
	hipMemcpy(d_freq, freq, N * sizeof(int), hipMemcpyHostToDevice);	

	parallelf <<< block_count , thread_count >>> (d_buffer , d_freq , file_size);

	hipMemcpy(freq, d_freq, N * sizeof(int), hipMemcpyDeviceToHost);

	for (j = 0; j < N; j++) {
		printf("%d = %d\n", j + base, freq[j]);
	}

	fclose(pFile);
	free(buffer);
	hipFree(d_buffer);
	hipFree(d_freq);

	return 0;
}
